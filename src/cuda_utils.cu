#include <hip/hip_runtime.h>
#include <iostream>

#include "cuda_utils.hpp"

void check_cuda_error(hipError_t err, const char *const func, const char *const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void check_last_cuda_error(const char *const file, const int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}